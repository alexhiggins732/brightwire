#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#define BLOCKSIZE 16
#define BLOCKSIZE2 BLOCKSIZE*BLOCKSIZE

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>

extern "C"
{
	/*const float TOO_SMALL = -1.0E20f;
	const float TOO_BIG = 1.0E20f;

	__global__ float _Constrain(float d)
	{
		if (isnan(d))
			return 0;
		else if (isinf(d))
			return TOO_BIG;
		else if (d < TOO_SMALL)
			return TOO_SMALL;
		else if (d > TOO_BIG)
			return TOO_BIG;
		return d;
	}*/

	__global__ void PointwiseMultiply(float* a, float* b, int size)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;

		if (index < size)
			b[index] *= a[index];
	}

	__global__ void PointwiseDivide(float* a, float* b, int size)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;

		if (index < size)
			b[index] = a[index] / b[index];
	}

	__global__ void Sqrt(float* a, float* b, int size, float valueAdjustment)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;

		if (index < size)
			b[index] = sqrt(a[index] + valueAdjustment);
	}

	__global__ void AddInPlace(float* a, float* b, int size, float coefficient1, float coefficient2)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;

		if (index < size)
			a[index] = (a[index] * coefficient1) + (b[index] * coefficient2);
	}

	__global__ void SubtractInPlace(float* a, float* b, int size, float coefficient1, float coefficient2)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;

		if (index < size)
			a[index] = (a[index] * coefficient1) - (b[index] * coefficient2);
	}

	__global__ void AddToEachRow(float* a, float* b, int rows, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		if (i < rows && j < columns)
			a[j * rows + i] += b[j];
	}

	__global__ void AddToEachColumn(float* a, float* b, int rows, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		if (i < rows && j < columns)
			a[j * rows + i] += b[i];
	}

	__global__ void TanH(float* a, float* b, int size)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;

		if (index < size)
			b[index] = tanh(a[index]);
	}

	__global__ void TanHDerivative(float* a, float* b, int size)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;

		if (index < size)
			b[index] = 1.0f - pow(tanh(a[index]), 2);
	}

	__global__ void Sigmoid(float* a, float* b, int size)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;

		if (index < size)
			b[index] = 1.0f / (1.0f + exp(-1.0f * a[index]));
	}

	__global__ void SigmoidDerivative(float* a, float* b, int size)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;

		if (index < size) {
			float sigmoid = 1.0f / (1.0f + exp(-1.0f * a[index]));
			b[index] = sigmoid * (1.0f - sigmoid);
		}
	}

	__global__ void RELU(float* a, float* b, int size)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;

		if (index < size) {
			float val = a[index];
			b[index] = (val <= 0) ? 0 : val;
		}
	}

	__global__ void RELUDerivative(float* a, float* b, int size)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;

		if (index < size) {
			float val = a[index];
			b[index] = (val <= 0) ? 0 : 1;
		}
	}

	__global__ void LeakyRELU(float* a, float* b, int size)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;

		if (index < size) {
			float val = a[index];
			b[index] = (val <= 0) ? 0.01f*val : val;
		}
	}

	__global__ void LeakyRELUDerivative(float* a, float* b, int size)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;

		if (index < size) {
			float val = a[index];
			b[index] = (val <= 0) ? 0.01f : 1;
		}
	}

	__global__ void Reverse(float* a, float* b, int size)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;

		if(index < size)
			b[size - index - 1] = a[index];
	}

	__global__ void SumRows(float* a, float* b, int rows, int columns)
	{
		int index = blockIdx.x * blockDim.x + threadIdx.x;

		if (index < rows) {
			float temp = 0;
			for (int i = 0; i < columns; i++)
				temp += a[i * rows + index];
			b[index] = temp;
		}
	}

	__global__ void SumColumns(float* a, float* b, int rows, int columns)
	{
		int index = blockIdx.x * blockDim.x + threadIdx.x;

		if (index < columns) {
			float temp = 0;
			for (int i = 0; i < rows; i++)
				temp += a[index * rows + i];
			b[index] = temp;
		}
	}

	__global__ void MemClear(float* data, int count, int srcOffset, int srcIncrement)
	{
		int index = blockIdx.x * blockDim.x + threadIdx.x;

		if (index < count) {
			int arrayIndex = srcOffset + (index * srcIncrement);
			data[arrayIndex] = 0.0f;
		}
	}

	__global__ void FindMinAndMax(float* data, int count, float* minBlock, float* maxBlock)
	{
		int tidX = threadIdx.x;
		int blockX = blockIdx.x;
		int index = blockDim.x * blockX + tidX;

		// read block into shared memory
		__shared__ float block[BLOCKSIZE2];
		block[tidX] = (index < count) ? data[index] : 0;
		__syncthreads();

		// aggregate per block
		if (tidX == 0) {
			float min = FLT_MAX, max = FLT_MIN;
			int maxIndex = BLOCKSIZE2;
			if (count - index < BLOCKSIZE2)
				maxIndex = count - index;
			for (int i = 0; i < maxIndex; i++) {
				float val = block[i];
				if (val > max)
					max = val;
				if (val < min)
					min = val;
			}
			minBlock[blockX] = min;
			maxBlock[blockX] = max;
		}
	}

	__global__ void FindSum(float* data, int count, float* sum)
	{
		int tidX = threadIdx.x;
		int blockX = blockIdx.x;
		int index = blockDim.x * blockX + tidX;

		// read block into shared memory
		__shared__ float block[BLOCKSIZE2];
		if (index < count)
			block[tidX] = data[index];
		__syncthreads();

		// aggregate per block
		if (tidX == 0) {
			float total = 0;
			int maxIndex = BLOCKSIZE2;
			if (count - blockX * BLOCKSIZE2 < BLOCKSIZE2)
				maxIndex = count - blockX * BLOCKSIZE2;
			for (int i = 0; i < maxIndex; i++) {
				total += block[i];
			}
			sum[blockX] = total;
		}
	}

	__global__ void FindStdDev(float* data, int count, float mean, float* stdDev)
	{
		int tidX = threadIdx.x;
		int blockX = blockIdx.x;
		int index = blockDim.x * blockX + tidX;

		// read block into shared memory
		__shared__ float block[BLOCKSIZE2];
		if (index < count)
			block[tidX] = data[index];
		__syncthreads();

		// aggregate per block
		if (tidX == 0) {
			float total = 0;
			int maxIndex = BLOCKSIZE2;
			if (count - blockX * BLOCKSIZE2 < BLOCKSIZE2)
				maxIndex = count - blockX * BLOCKSIZE2;
			for (int i = 0; i < maxIndex; i++) {
				total += pow(block[i] - mean, 2);
			}
			stdDev[blockX] = total;
		}
	}

	__global__ void Constrain(float* data, int count, float min, float max)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < count) {
			float val = data[i];
			if (val < min)
				data[i] = min;
			if (val > max)
				data[i] = max;
		}
	}

	__global__ void Pow(float* a, float* b, int count, float power)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < count) {
			float val = a[i];
			b[i] = pow(val, power);
		}
	}

	__global__ void Diagonal(float* a, float* b, int rows, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < rows && i < columns)
			b[i] = a[i * rows + i];
	}

	__global__ void L1Regularisation(float* a, int count, float coefficient)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < count) {
			float val = a[i];
			a[i] = val - ((val > 0 ? 1 : val < 0 ? -1 : 0) * coefficient);
		}
	}

	__global__ void PointwiseDivideRows(float* a, float* b, int rows, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;

		if (i < rows && j < columns) {
			int index = j * rows + i;
			float val = a[index];
			a[index] = val / b[i];
		}
	}

	__global__ void PointwiseDivideColumns(float* a, float* b, int rows, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;

		if (i < rows && j < columns) {
			int index = j * rows + i;
			float val = a[index];
			a[index] = val / b[j];
		}
	}

	__global__ void SplitRows(float* a, float* b, float* c, int rows, int columns, int position)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;

		if (i < rows && j < columns) {
			int index = j * rows + i;
			float val = a[index];
			if(j >= position) {
				int diff = j - position;
				c[diff * rows + i] = val;
			}else
				b[index] = val;
		}
	}

	__global__ void SplitColumns(float* a, float* b, float* c, int rows, int columns, int position)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;

		if (i < rows && j < columns) {
			float val = a[j * rows + i];
			if(i >= position) {
				int diff = i - position;
				c[j * (rows-position) + diff] = val;
			}else
				b[j * position + i] = val;
		}
	}

	__global__ void ConcatColumns(float* a, float* b, float* c, int rows, int columns, int topRowCount, int bottomRowCount)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;

		if (i < rows && j < columns) {
			float val;
			if(i >= topRowCount)
				val = b[j * bottomRowCount + i - topRowCount];
			else
				val = a[j * topRowCount + i];
			c[j * rows + i] = val;
		}
	}

	__global__ void ConcatRows(float* a, float* b, float* c, int rows, int columns, int leftColumnCount)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;

		if (i < rows && j < columns) {
			float val;
			if(j >= leftColumnCount)
				val = b[(j-leftColumnCount) * rows + i];
			else
				val = a[j * rows + i];
			c[j * rows + i] = val;
		}
	}

	__global__ void EuclideanDistance(float* a, float* b, float* c, int count)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < count)
			c[i] = pow(a[i] - b[i], 2);
	}

	__global__ void MultiEuclideanDistance(float* a, float** b, float* c, int size, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;

		if (i < size && j < columns) {
			float val1 = a[i];
			float val2 = b[j][i];
			c[j * size + i] = pow(val1 - val2, 2);
		}
	}

	__global__ void ManhattanDistance(float* a, float* b, float* c, int count)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < count)
			c[i] = abs(a[i] - b[i]);
	}

	__global__ void MultiManhattanDistance(float* a, float** b, float* c, int size, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;

		if (i < size && j < columns) {
			float val1 = a[i];
			float val2 = b[j][i];
			c[j * size + i] = abs(val1 - val2);
		}
	}

	__global__ void Abs(float* a, float* b, int count)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < count) {
			b[i] = abs(a[i]);
		}
	}

	__global__ void Log(float* a, float* b, int count)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < count)
			b[i] = log(a[i]);
	}

	__global__ void Normalise(float* a, int count, float min, float range)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < count)
			a[i] = (a[i] - min) / range;
	}

	__global__ void SoftmaxVector(float* a, float* b, int count, float max)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < count)
			b[i] = exp(a[i] - max);
	}

	__global__ void VectorAdd(float* a, int size, float scalar)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < size)
			a[i] += scalar;
	}

	__global__ void VectorCopyRandom(float* a, float* b, int* c, int size)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < size) {
			int index = c[i];
			b[i] += a[index];
		}
	}

	__global__ void CopyToMatrix(float** a, float* b, int rows, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;

		if (i < rows && j < columns) {
			float val = a[i][j];
			b[j * rows + i] = val;
		}
	}

	__global__ void VectorSplit(float* a, float** b, int inputSize, int blockSize)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < inputSize) {
			int offset = i / blockSize;
			int index = i % blockSize;
			float val = a[i];
			b[offset][index] = val;
		}
	}

	__global__ void TensorConvertToVector(float** a, float* b, int matrixSize, int size)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < size) {
			int offset = i / matrixSize;
			int index = i % matrixSize;
			b[i] = a[offset][index];
		}
	}

	__global__ void TensorConvertToMatrix(float** a, float* b, int aRows, int aColumns, int bRows, int bColumns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;

		if (i < bRows && j < bColumns) {
			//int x = i / aColumns;
			//int y = i % aColumns;
			int x = i % aRows;
			int y = i / aRows;
			b[j * bRows + i] = a[j][y * aRows + x];
		}
	}

	__global__ void TensorAddPadding(float*** a, float*** b, int count, int aRows, int aColumns, int bRows, int bColumns, int depth, int padding)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		int size = bRows * depth;
		int z = index / size;
		int index2 = index % size;
		int k = index2 / bRows;
		int i = index2 % bRows;

		if (z < count && k < depth && i >= padding && j >= padding && i < bRows-padding && j < bColumns-padding) {
			int aIndex = (j-padding) * aRows + (i-padding);
			int bIndex = j * bRows + i;
			b[z][k][bIndex] = a[z][k][aIndex];
		}
	}

	__global__ void TensorRemovePadding(float*** a, float*** b, int count, int aRows, int aColumns, int bRows, int bColumns, int depth, int padding)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		int size = aRows * depth;
		int z = index / size;
		int index2 = index % size;
		int k = index2 / aRows;
		int i = index2 % aRows;

		if (z < count && k < depth && j >= padding && i >= padding && i < aRows-padding && j < aColumns-padding) {
			int aIndex = j * aRows + i;
			int bIndex = (j-padding) * bRows + (i-padding);
			b[z][k][bIndex] = a[z][k][aIndex];
		}
	}

	__global__ void TensorIm2Col(float*** a, float** b, int count, int aRows, int aColumns, int bRows, int bColumns, int depth, int filterWidth, int filterHeight, int stride)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		int z = index / bRows;
		int i = index % bRows;

		if (z < count && j < bColumns) {
			int blockSize = filterWidth * filterHeight;
			int depthIndex = j / blockSize;
			int localIndex = j % blockSize;
			float* channel = a[z][depthIndex];

			int xExtent = (aColumns - filterWidth) / stride + 1;
			int yExtent = (aRows - filterHeight) / stride + 1;
			int xOffset = i / xExtent * stride;
			int yOffset = i % yExtent * stride;
			
			int ax = xOffset + (localIndex / filterWidth);
			int ay = yOffset + (localIndex % filterHeight);

			float val = channel[ax * aRows + ay];
			b[z][j * bRows + i] = val;
		}
	}

	__global__ void TensorReverseIm2Col(float*** a, float*** b, float*** c, int count, int aRows, int aColumns, int depth, int cRows, int cSize, int inputDepth, int filterHeight, int filterWidth, int stride)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;

		int mx = aRows * depth;
		int mx2 = mx * inputDepth;
		int z = index / mx2;
		int index2 = index % mx2;

		int k2 = index2 / mx;
		int index3 = index2 % mx;
		int k = index3 / aRows;
		int i = index3 % aRows;

		if(z < count && k < depth && k2 < inputDepth && j < aColumns) {
			int x1 = j*stride;
			int y1 = i*stride;
			float* slice = a[z][k];
			float** filterList = b[k];
			float* output = c[z][k];
			float* filter = filterList[k2];

			float error = slice[i * aRows + j];
			for (int fx = 0; fx < filterWidth; fx++) {
				for (int fy = 0; fy < filterHeight; fy++) {
					int cx = fx + x1;
					int cy = fy + y1;
					int filterIndex = fx * filterHeight + fy;
					int outputRow = cx * cRows + cy;
					output[k2 * cSize + outputRow] = filter[filterIndex] * error;
				}
			}
		}
	}

	__global__ void SoftmaxDerivative(float* a, float* b, int size)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;

		if(i < size && j < size) {
			int index = j * size + i;
			if(i == j)
				b[index] = a[i] * (1 - a[i]);
			else
				b[index] = -a[i] * a[j];
		}
	}

	__global__ void Rotate(float** a, float* b, int size, int blockCount, int blockSize)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;
		
		if(index < size) {
			int blockIndex = index / blockCount;
			int blockSubIndex = index % blockSize;
			b[index] = a[blockCount - blockIndex - 1][blockSize - blockSubIndex - 1];
		}
	}

	__global__ void TensorMaxPool(float** a, float** b, int** bestXIndexPtr, int** bestYIndexPtr, int aRows, int aColumns, int depth, int bRows, int bColumns, int filterWidth, int filterHeight, int stride)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		int z = index / bRows;
		int i = index % bRows;

		if(z < depth && j < bColumns) {
			int index = j * bRows + i;
			int aX = j * stride;
			int aY = i * stride;

			float* source = a[z];
			float* target = b[z];
			int* bestXIndex = bestXIndexPtr[z];
			int* bestYIndex = bestYIndexPtr[z];
	
			float maxVal = FLT_MIN;
			int bestX = -1;
			int bestY = -1;
			for (int fx = 0; fx < filterWidth; fx++) {
				for (int fy = 0; fy < filterHeight; fy++) {
					int xPos = aX + fx;
					int yPos = aY + fy;
					float val = source[xPos * aRows + yPos];
					if (val > maxVal || bestX == -1) {
						bestX = xPos;
						bestY = yPos;
						maxVal = val;
					}
				}
			}
			if(bestXIndex) {
				bestXIndex[index] = bestX;
			}
			if(bestYIndex) {
				bestYIndex[index] = bestY;
			}
			target[index] = maxVal;
		}
	}

	__global__ void TensorReverseMaxPool(float** a, float** b, int** bestXIndexPtr, int** bestYIndexPtr, int aRows, int aColumns, int depth, int bRows, int bColumns)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		int z = index / aRows;
		int i = index % aRows;

		if(z < depth && j < aColumns) {
			int index = j * aRows + i;
			float* source = a[z];
			float* target = b[z];
			int* bestXIndex = bestXIndexPtr[z];
			int* bestYIndex = bestYIndexPtr[z];
			int targetX = bestXIndex[index];
			int targetY = bestYIndex[index];
			target[targetX * bRows + targetY] = source[index];
		}
	}
}