#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>

#define BLOCKSIZE 16
#define BLOCKSIZE2 BLOCKSIZE*BLOCKSIZE

extern "C"
{
    __global__ void IsFinite(float* a, float* b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] = isfinite(a[index]) ? 0 : 1;
        }
	}

	__global__ void PointwiseMultiply(float* a, float* b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] *= a[index];
        }
	}

	__global__ void PointwiseDivide(float* a, float* b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] = a[index] / b[index];
        }
	}

	__global__ void Sqrt(float* a, float* b, int size, float valueAdjustment)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] = sqrt(a[index] + valueAdjustment);
        }
	}

	__global__ void AddInPlace(float* a, float* b, int size, float coefficient1, float coefficient2)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            a[index] = (a[index] * coefficient1) + (b[index] * coefficient2);
        }
	}

	__global__ void SubtractInPlace(float* a, float* b, int size, float coefficient1, float coefficient2)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            a[index] = (a[index] * coefficient1) - (b[index] * coefficient2);
        }
	}

	__global__ void AddToEachRow(float* a, float* b, int rows, int columns)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                a[j * rows + i] += b[j];
            }
        }
	}

	__global__ void AddToEachColumn(float* a, float* b, int rows, int columns)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                a[j * rows + i] += b[i];
            }
        }
	}

	__global__ void TanH(float* a, float* b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] = tanh(a[index]);
        }
	}

	__global__ void TanHDerivative(float* a, float* b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] = 1.0f - pow(tanh(a[index]), 2);
        }
	}

	__global__ void Sigmoid(float* a, float* b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] = 1.0f / (1.0f + exp(-1.0f * a[index]));
        }
	}

	__global__ void SigmoidDerivative(float* a, float* b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float sigmoid = 1.0f / (1.0f + exp(-1.0f * a[index]));
			b[index] = sigmoid * (1.0f - sigmoid);
        }
	}

	__global__ void RELU(float* a, float* b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float val = a[index];
			b[index] = (val <= 0) ? 0 : val;
        }
	}

	__global__ void RELUDerivative(float* a, float* b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float val = a[index];
			b[index] = (val <= 0) ? 0 : 1;
        }
	}

	__global__ void LeakyRELU(float* a, float* b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float val = a[index];
			b[index] = (val <= 0) ? 0.01f*val : val;
        }
	}

	__global__ void LeakyRELUDerivative(float* a, float* b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float val = a[index];
			b[index] = (val <= 0) ? 0.01f : 1;
        }
	}

	__global__ void Reverse(float* a, float* b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[size - index - 1] = a[index];
        }
	}

	__global__ void SumRows(float* a, float* b, int rows, int columns)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < rows; index += blockDim.x * gridDim.x) {
            float temp = 0;
			for (int i = 0; i < columns; i++)
				temp += a[i * rows + index];
			b[index] = temp;
        }
	}

	__global__ void SumColumns(float* a, float* b, int rows, int columns)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < columns; index += blockDim.x * gridDim.x) {
            float temp = 0;
			for (int i = 0; i < rows; i++)
				temp += a[index * rows + i];
			b[index] = temp;
        }
	}

	__global__ void MemClear(float* data, int count, int srcOffset, int srcIncrement)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            int arrayIndex = srcOffset + (index * srcIncrement);
			data[arrayIndex] = 0.0f;
        }
	}

	__global__ void FindMinAndMax(float* data, int count, float* minBlock, float* maxBlock)
	{
		int tidX = threadIdx.x;
		int blockX = blockIdx.x;
		int index = blockDim.x * blockX + tidX;

		// read block into shared memory
		__shared__ float block[BLOCKSIZE2];
		block[tidX] = (index < count) ? data[index] : 0;
		__syncthreads();

		// aggregate per block
		if (tidX == 0) {
			float min = FLT_MAX, max = FLT_MIN;
			int maxIndex = BLOCKSIZE2;
			if (count - index < BLOCKSIZE2)
				maxIndex = count - index;
			for (int i = 0; i < maxIndex; i++) {
				float val = block[i];
				if (i == 0 || val > max)
					max = val;
				if (i == 0 || val < min)
					min = val;
			}
			minBlock[blockX] = min;
			maxBlock[blockX] = max;
		}
	}

	__global__ void FindSum(float* data, int count, float* sum)
	{
		int tidX = threadIdx.x;
		int blockX = blockIdx.x;
		int index = blockDim.x * blockX + tidX;

		// read block into shared memory
		__shared__ float block[BLOCKSIZE2];
		if (index < count)
			block[tidX] = data[index];
		__syncthreads();

		// aggregate per block
		if (tidX == 0) {
			float total = 0;
			int maxIndex = BLOCKSIZE2;
			if (count - blockX * BLOCKSIZE2 < BLOCKSIZE2)
				maxIndex = count - blockX * BLOCKSIZE2;
			for (int i = 0; i < maxIndex; i++) {
				total += block[i];
			}
			sum[blockX] = total;
		}
	}

	__global__ void FindStdDev(float* data, int count, float mean, float* stdDev)
	{
		int tidX = threadIdx.x;
		int blockX = blockIdx.x;
		int index = blockDim.x * blockX + tidX;

		// read block into shared memory
		__shared__ float block[BLOCKSIZE2];
		if (index < count)
			block[tidX] = data[index];
		__syncthreads();

		// aggregate per block
		if (tidX == 0) {
			float total = 0;
			int maxIndex = BLOCKSIZE2;
			if (count - blockX * BLOCKSIZE2 < BLOCKSIZE2)
				maxIndex = count - blockX * BLOCKSIZE2;
			for (int i = 0; i < maxIndex; i++) {
				total += pow(block[i] - mean, 2);
			}
			stdDev[blockX] = total;
		}
	}

	__global__ void Constrain(float* data, int count, float min, float max)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            float val = data[index];
			if (val < min)
				data[index] = min;
			if (val > max)
				data[index] = max;
        }
	}

	__global__ void Pow(float* a, float* b, int count, float power)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            float val = a[index];
			b[index] = pow(val, power);
        }
	}

	__global__ void Diagonal(float* a, float* b, int rows, int columns)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < rows && index < columns; index += blockDim.x * gridDim.x) {
            b[index] = a[index * rows + index];
        }
	}

	__global__ void L1Regularisation(float* a, int count, float coefficient)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            float val = a[index];
			a[index] = val - ((val > 0 ? 1 : val < 0 ? -1 : 0) * coefficient);
        }
	}

	__global__ void PointwiseDivideRows(float* a, float* b, int rows, int columns)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                int index = j * rows + i;
			    float val = a[index];
			    a[index] = val / b[i];
            }
        }
	}

	__global__ void PointwiseDivideColumns(float* a, float* b, int rows, int columns)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                int index = j * rows + i;
			    float val = a[index];
			    a[index] = val / b[j];
            }
        }
	}

	__global__ void SplitRows(float* a, float* b, float* c, int rows, int columns, int position)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                int index = j * rows + i;
			    float val = a[index];
			    if(j >= position) {
				    int diff = j - position;
				    c[diff * rows + i] = val;
			    }else
				    b[index] = val;
            }
        }
	}

	__global__ void SplitColumns(float* a, float* b, float* c, int rows, int columns, int position)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val = a[j * rows + i];
			    if(i >= position) {
				    int diff = i - position;
				    c[j * (rows-position) + diff] = val;
			    }else
				    b[j * position + i] = val;
            }
        }
	}

	__global__ void ConcatColumns(float* a, float* b, float* c, int rows, int columns, int topRowCount, int bottomRowCount)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val;
			    if(i >= topRowCount)
				    val = b[j * bottomRowCount + i - topRowCount];
			    else
				    val = a[j * topRowCount + i];
			    c[j * rows + i] = val;
            }
        }
	}

	__global__ void ConcatRows(float* a, float* b, float* c, int rows, int columns, int leftColumnCount)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val;
			    if(j >= leftColumnCount)
				    val = b[(j-leftColumnCount) * rows + i];
			    else
				    val = a[j * rows + i];
			    c[j * rows + i] = val;
            }
        }
	}

	__global__ void EuclideanDistance(float* a, float* b, float* c, int count)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            c[index] = pow(a[index] - b[index], 2);
        }
	}

	__global__ void MultiEuclideanDistance(float* a, float** b, float* c, int size, int columns)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val1 = a[i];
			    float val2 = b[j][i];
			    c[j * size + i] = pow(val1 - val2, 2);
            }
        }
	}

	__global__ void ManhattanDistance(float* a, float* b, float* c, int count)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            c[index] = abs(a[index] - b[index]);
        }
	}

	__global__ void MultiManhattanDistance(float* a, float** b, float* c, int size, int columns)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val1 = a[i];
			    float val2 = b[j][i];
			    c[j * size + i] = abs(val1 - val2);
            }
        }
	}

	__global__ void Abs(float* a, float* b, int count)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            b[index] = abs(a[index]);
        }
	}

	__global__ void Log(float* a, float* b, int count)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            b[index] = log(a[index]);
        }
	}

	__global__ void Normalise(float* a, int count, float min, float range)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            a[index] = (a[index] - min) / range;
        }
	}

	__global__ void SoftmaxVector(float* a, float* b, int count, float max)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            b[index] = exp(a[index] - max);
        }
	}

	__global__ void VectorAdd(float* a, int size, float scalar)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            a[index] += scalar;
        }
	}

	__global__ void VectorCopyRandom(float* a, float* b, int* c, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
			b[index] += a[c[index]];
        }
	}

	__global__ void CopyToMatrixRows(float** a, float* b, int rows, int columns)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val = a[i][j];
			    b[j * rows + i] = val;
            }
        }
	}

    __global__ void CopyToMatrixColumns(float** a, float* b, int rows, int columns)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val = a[j][i];
                //printf("i:%i(%i) j:%i(%i)\n", i, rows, j, columns);
			    b[j * rows + i] = val;
            }
        }
	}

	/*__global__ void VectorSplit(float* a, float** b, int inputSize, int blockSize)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < inputSize; index += blockDim.x * gridDim.x) {
            b[index / blockSize][index % blockSize] = a[index];
        }
	}

	__global__ void TensorConvertToVector(float** a, float* b, int matrixSize, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            int offset = index / matrixSize;
			int index2 = index % matrixSize;
			b[index] = a[offset][index2];
        }
	}

	__global__ void TensorConvertToMatrix(float** a, float* b, int aRows, int aColumns, int bRows, int bColumns)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < bRows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < bColumns; j += blockDim.y * gridDim.y) {
                int x = i % aRows;
			    int y = i / aRows;
			    b[j * bRows + i] = a[j][y * aRows + x];
            }
        }
	}*/

	__global__ void TensorAddPadding(
        int size, 
        float* a, 
        float* b, 
        int rows, 
        int columns, 
        int depth, 
        int count, 
        int outputRows, 
        int outputColumns, 
        int padding
    ) {
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            int i = index % outputRows;
            int index2 = index / outputRows;

            int j = index2 % outputColumns;
            int index3 = index2 / outputColumns;

            int k = index3 % depth;
            int z = index3 / depth;

            float val = 0;
            if(i >= padding && i < (outputRows - padding) && j >= padding && j < (outputColumns - padding)) {
                float* inputPtr = a + (rows * columns * depth * z) + (rows * columns * k);
                int aIndex = (j-padding) * rows + (i-padding);
                val = inputPtr[aIndex];

                /*printf("index:%i i:%i(%i) j:%i(%i) k:%i(%i) z:%i(%i) ai:%i val:%f\n", index,
                    i, outputRows,
                    j, outputColumns, 
                    k, depth, 
                    z, count,
                    aIndex, val
                );*/
            }

            float* outputPtr = b + (outputRows * outputColumns * depth * z) + (outputRows * outputColumns * k);
            outputPtr[j * outputRows + i] = val;
        }
	}

	__global__ void TensorRemovePadding(
        int size, 
        float* a, 
        float* b, 
        int rows, 
        int columns, 
        int depth, 
        int count, 
        int outputRows, 
        int outputColumns, 
        int padding
    ) {
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            int i = index % rows;
            int index2 = index / rows;

            int j = index2 % columns;
            if(i >= padding && i < (rows-padding) && j >= padding && j < (columns-padding)) {
                int index3 = index2 / columns;

                int k = index3 % depth;
                int z = index3 / depth;

                float* inputPtr = a + (rows * columns * depth * z) + (rows * columns * k);
                int aIndex = j * rows + i;
                float val = inputPtr[aIndex];

                float* outputPtr = b + (outputRows * outputColumns * depth * z) + (outputRows * outputColumns * k);
                int bIndex = (j-padding) * outputRows + (i-padding);
                outputPtr[bIndex] = val;

                /*printf("index:%i i:%i(%i) j:%i(%i) k:%i(%i) z:%i(%i) ai:%i bi:%i val:%f\n", index,
                    i, outputRows,
                    j, outputColumns, 
                    k, depth, 
                    z, count,
                    aIndex, bIndex,
                    val
                );*/
            }
        }
	}

    __global__ void TensorIm2Col(
        int size, 
        float* a, 
        float* b, 
        float* cx, 
        float* cy, 
        int rows,
        int columns,
        int depth,
        int count,
        int outputRows,
        int outputColumns,
        int convolutionCount, 
        int filterWidth, 
        int filterHeight,
        int stride
    ) {
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            int x = index % filterWidth;
            int index2 = index / filterWidth;

            int y = index2 % filterHeight;
            int index3 = index2 / filterHeight;

            int k = index3 % depth;
            int index4 = index3 / depth;

            int ci = index4 % convolutionCount;
            int i = index4 / convolutionCount;
            
			//int extent = (rows - filterWidth) / stride + 1;
            //int offsetY = ci / extent * stride;
            //int offsetX = ci % extent * stride;

            int offsetX = cx[ci];
            int offsetY = cy[ci];

            /*printf("index:%i, i:%i(%i), ci:%i(%i), k:%i(%i), x:%i(%i), y:%i(%i), cx:%i=%i, cy:%i=%i\n", index,
                i, count,
                ci, convolutionCount,
                k, depth,
                x, filterWidth,
                y, filterHeight,
                offsetX, (int)cx[ci],
                offsetY, (int)cy[ci]
            );*/

            int filterOffset = k * filterWidth * filterHeight;
            int filterIndex = filterOffset + (x * filterHeight + y);

            float* outputPtr = b + (outputRows * outputColumns * i);
            float* inputPtr = a + (rows * columns * depth * i) + (rows * columns * k);
            outputPtr[filterIndex * outputRows + ci] = inputPtr[(offsetX + x) * rows + (offsetY + y)];
        }
    }

    __global__ void TensorReverseIm2Col(
        int size, 
        float* a, 
        float* filters, 
        float* b, 
        float* cx, 
        float* cy, 
        int rows, 
        int columns, 
        int depth, 
        int count,
        int convolutionCount,  
        int filterWidth, 
        int filterHeight, 
        int stride, 
        int outputRows,
        int outputColumns,
        int outputDepth
    ) {
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            int z = index % outputDepth;
            int index2 = index / outputDepth;

            int x = index2 % filterWidth;
            int index3 = index2 / filterWidth;

            int y = index3 % filterHeight;
            int index4 = index3 / filterHeight;

            int ci = index4 % convolutionCount;
            int index5 = index4 / convolutionCount;

            int k = index5 % depth;
            int i = index5 / depth;

            int offsetX = cx[ci];
            int offsetY = cy[ci];

            /*printf("index:%i di:%i(%i) ci:%i(%i) k:%i(%i) x:%i(%i) y:%i(%i) z:%i(%i) cx:%i cy:%i\n", index,
                i, count,
                ci, convolutionCount, 
                k, depth, 
                x, filterWidth, 
                y, filterHeight, 
                z, outputDepth, 
                offsetX, offsetY
            );*/

            float* slice = a + (i * rows * columns * depth) + (k * rows * columns);
            float* filter = filters + (k * outputDepth * filterWidth * filterHeight) + (z * filterWidth * filterHeight);
            float* output = b + (k * outputRows * outputColumns * outputDepth * count) 
                + (i * outputRows * outputColumns * outputDepth) 
                + (z * outputRows * outputColumns)
            ;

            int errorX = offsetX / stride;
            int errorY = offsetY / stride;
            float error = slice[errorX * rows + errorY];

            int filterIndex = x * filterHeight + y;
            int outputIndex = (offsetX+x) * outputRows + (offsetY+y);
            float val = filter[filterIndex] * error;

            output[outputIndex] = val;
        }
    }

	__global__ void SoftmaxDerivative(float* a, float* b, int size)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < size; j += blockDim.y * gridDim.y) {
                int index = j * size + i;
			    if(i == j)
				    b[index] = a[i] * (1 - a[i]);
			    else
				    b[index] = -a[i] * a[j];
            }
        }
	}

	__global__ void RotateInPlace(float* a, int size, int blockCount, int blockSize)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            int blockIndex = index / blockSize;
			int blockOffset = index % blockSize;
            int index1 = blockIndex * blockSize + blockSize - blockOffset - 1;
			int index2 = blockIndex * blockSize + blockOffset; 
			float temp = a[index1];
			a[index1] = a[index2];
			a[index2] = temp;
        }
	}

	__global__ void TensorMaxPool(
        int size, 
        float* a, 
        float* b, 
        float* indexOffset,
        float* cx, 
        float* cy,
        int convolutionCount,
        int rows, 
        int columns, 
        int depth, 
        int count, 
        int outputRows, 
        int outputColumns, 
        int filterWidth, 
        int filterHeight, 
        int stride,
        int saveIndices
    ) {
		for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            int ci = index % convolutionCount;
            int index2 = index / convolutionCount;

            int k = index2 % depth;
            int z = index2 / depth;

            int aX = cx[ci];
			int aY = cy[ci];
            int bX = aX / stride;
            int bY = aY / stride;

            /*printf("index:%i k:%i(%i) z:%i(%i) ax:%i ay:%i bx:%i by:%i\n", index,
                k, depth, 
                z, count,
                aX, aY,
                bX, bY
            );*/

            int targetOffset = (z * outputRows * outputColumns * depth) + (k * outputRows * outputColumns);
            float* source = a + (z * rows * columns * depth) + (k * rows * columns);
            float* target = b + targetOffset;

            float maxVal = 0;
	        int bestOffset = -1;
	        int offset = 0;
	                
	        for (int x = 0; x < filterWidth; x++) {
		        for (int y = 0; y < filterHeight; y++) {
			        float val = source[(aX + x) * rows + (aY + y)];
                    bool isGreater = (bestOffset < 0 || val > maxVal);
			        if (isGreater) {
				        bestOffset = offset;
				        maxVal = val;
			        }
                    //printf("index:%i, x:%i, y:%i val:%f max:%f offset:%i is-greater:%i\n", index, x, y, val, maxVal, bestOffset, isGreater ? 1 : 0);
					++offset;
		        }
	        }

            //printf("\tindex:%i i:%i j:%i val:%f\n", index, i, j, maxVal);
            if(saveIndices) {
                float* indices = indexOffset + targetOffset;
                indices[bX * outputRows + bY] = bestOffset;
            }
            target[bX * outputRows + bY] = maxVal;
        }
	}

	__global__ void TensorReverseMaxPool(
        int size, 
        float* a,
        float* indices,
        float* b, 
        int rows,
        int columns,
        int depth,
        int count,
        int outputRows,
        int outputColumns,
        int filterWidth,
        int filterHeight,
        int stride
    ) {
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            int i = index % rows;
            int index2 = index / rows;

            int j = index2 % columns;
            int index3 = index2 / columns;

            int k = index3 % depth;
            int z = index3 / depth;

            int sourceOffset = (z * rows * columns * depth) + (k * rows * columns);
            float* source = a + sourceOffset;
            float* indexPtr = indices + sourceOffset;
            float* target = b + (z * outputRows * outputColumns * depth) + (k * outputRows * outputColumns);
            int sourceIndex = j * rows + i;
            float val = source[sourceIndex];
            int offset = indexPtr[sourceIndex];

            if(offset < 0)
                offset = 0;

            int targetX = j * stride + (offset / filterHeight);
            int targetY = i * stride + (offset % filterHeight);

            /*printf("index:%i s:%i i:%i(%i) j:%i(%i) k:%i(%i) z:%i(%i) val:%f offset:%i tx:%i ty:%i\n", 
                index, stride,
                i, outputRows,
                j, outputColumns, 
                k, depth, 
                z, count,
                val, offset,
                targetX, targetY
            );*/

            target[targetX * outputRows + targetY] = val;
        }
	}
}